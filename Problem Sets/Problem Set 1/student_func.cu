#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

#define NUM_THREADS 32

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < numCols && y < numRows) {
    int grayScaleOffset = y * numCols + x;
    int rgbOffset = grayScaleOffset;

    float R = rgbaImage[rgbOffset].x;
    float G = rgbaImage[rgbOffset].y;
    float B = rgbaImage[rgbOffset].z;
    float I = .299f * R + .587f * G + .114f * B;

    greyImage[grayScaleOffset] = I;
  }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols) {
  
  const dim3 gridSize(ceil((float)numCols/NUM_THREADS), ceil((float)numRows/NUM_THREADS), 1);  
  const dim3 blockSize(numCols, numRows, 1); 
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
